#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"

typedef at::BFloat16 dtype;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               float *__restrict__ _state, const F *__restrict__ const _r, const F *__restrict__ const _w, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ const _a, const F *__restrict__ const _b,
                               F *__restrict__ const _y)
{
    const int bbb = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _state += bbb*C*_N_ + h*_N_*_N_ + i*_N_;

    float state[_N_];
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        state[j] = _state[j];

    __shared__ float r[_N_], k[_N_], w[_N_], a[_N_], b[_N_];

    for (int _t = 0; _t < T; _t++)
    {
        const int t = bbb*T*C + h*_N_ + i + _t * C;
        __syncthreads();
        r[i] = float(_r[t]);
        w[i] = __expf(-__expf(float(_w[t])));
        k[i] = float(_k[t]);
        a[i] = float(_a[t]);
        b[i] = float(_b[t]);
        __syncthreads();

        float sa = 0;
        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            sa += a[j] * state[j];
        }

        float vv = float(_v[t]);
        float y = 0;
        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float& s = state[j];
            s = s * w[j] + k[j] * vv + sa * b[j];
            y += s * r[j];
        }
        _y[t] = F(y);
    }
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        _state[j] = state[j];    
}

void cuda_forward(int B, int T, int C, int H, float *state, dtype *r, dtype* w, dtype *k, dtype *v, dtype *a, dtype *b, dtype *y)
{
    assert(H*_N_ == C);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, state, r, w, k, v, a, b, y);
}
